#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <chrono>

hipError_t addWithCuda(int *c, const int *a, unsigned int size);

using namespace std;

/*__device__ bool Prime(long long n)
{
	for (int i = 2; i <= sqrt((double)n); i++)
		if (n%i == 0)
			return false;
	return true;
}*/

__global__ void addKernel(char *output, long long from, int *a, int cudaCores)
{
	long long current = threadIdx.x + from + cudaCores * blockIdx.x;

	long long outPos = current - from;

	output[outPos] = 0;

	if (a[current] % current == 0) output[outPos] = 1;
	else output[outPos] = -1;
}

int main()
{
	const int arraySize = 10000000;
	int *a = new int[arraySize];
	int *c = new int[arraySize];

	for (int c = 1; c < arraySize; c++)
	{
		a[c - 1] = c;
	}

	// Add vectors in parallel.
	auto begin = chrono::high_resolution_clock::now();
	hipError_t cudaStatus = addWithCuda(c, a, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	auto end = chrono::high_resolution_clock::now();

	/*int i = 0;
	while (c[i] > 0)
	{
		cout << c[i] << endl;
		i++;
	}*/

	cout << "Work time: " << chrono::duration_cast<chrono::milliseconds>(end - begin).count() << endl;
	system("pause");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, unsigned int size)
{
	int cudaCores = 1000;

	long long from = 2;
	const long long bufferSize = size - from;
	const long long blockCount = (bufferSize / cudaCores) + (bufferSize%cudaCores == 0 ? 0 : 1);

	if (bufferSize < cudaCores)
	{
		cudaCores = bufferSize;
	}

	char *output = new char[bufferSize];
	char *dev_output;

	int *dev_a = 0;
	//int *dev_c = 0;
	hipError_t cudaStatus;
	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_output, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//int threadsPerBlock = 55;
	//int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	// Launch a kernel on the GPU with one thread for each element.

	hipEventRecord(start, 0);

	addKernel << < blockCount, cudaCores >> > (dev_output, from, dev_a, cudaCores);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	hipEventRecord(stop, 0);
	float time = 0;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_output, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_output);
	hipFree(dev_a);

	/*int l = 0;
	while (l < bufferSize)
	{
		cout << c[l] << endl;
		l++;
	}*/

	/*int y = 0;
	while (y < bufferSize)
	{
		if ((int)output[y] > 0) { cout << "Chislo " << output[y] << " ne prostoe" << endl; break; }
		y++;
	}*/

	cout << "Work time: " << time << endl;
	system("pause");

	return cudaStatus;
}